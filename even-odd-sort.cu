#include <stdio.h>
#include <hip/hip_runtime.h>

//Device functions can only be called from other device or global functions. __device__ functions cannot be called from host code.

//Global functions are also called "kernels". It's the functions that you may call from the host side using CUDA kernel call semantics (<<<...>>>).
__global__ void testKernel(int *in, int *out, int size){

	bool oddeven = true;
	//__shared__ for shared memory
	__shared__ bool swappedodd;
	__shared__ bool swappedeven;
	
	int temp;

	while(1){
	
		if(oddeven == true){
			
/*
Using __syncthreads is sometimes necessary to ensure that all data from all threads is valid before threads read from shared memory which is written to by other threads.
			
__syncthreads() waits until all threads within the same block has reached the command and all threads within a warp (group of 32 threads) - that means all warps that belongs to a threadblock must reach the statement.
			
CUDA model is MIMD but current NVIDIA GPUs implement __syncthreads() at warp granularity instead of thread. It means, these are warps inside a thread-block who are synchronized not necessarily
*/
			__syncthreads();

			swappedeven=false;

			__syncthreads();

			//first column only which would have the array
			if (threadIdx.y == 0) {
			
				int idx = threadIdx.x;
				
				//0, 1, 2 threads will go through
				if( idx < (size/2) ){
					//COMPARISONS:
					// 0 <--> 1
					// 2 <--> 3
					// 4 <--> 5
					if ( in[2*idx] > in[2*idx+1] ){
						//BUBBLE SORT LOGIC
						temp= in[2*idx];
						in[2*idx]=in[2*idx+1];
						in[2*idx+1]=temp;
						swappedeven=true;
					
					}
				}
			}
			__syncthreads();
		}
		else{

			__syncthreads();

			swappedodd=false;

			__syncthreads();

			if (threadIdx.y == 0) {

				int idx = threadIdx.x;
				//0, 1 will go through
				if( idx < (size/2)-1 ){
					//COMPARISONS:
					// 1 <--> 2
					// 3 <--> 4
					if ( in[2*idx+1] > in[2*idx+2] ){

						temp=in[2*idx+1];
						in[2*idx+1]=in[2*idx+2];
						in[2*idx+2]=temp;
						swappedodd=true;

					}

				}


			}

			__syncthreads();

		}
	
	//if there are no swaps in odd phase as well as even phase then break (which means all sorting is done)
	// !(false) => true
	if( !( swappedodd || swappedeven ) )
		break;

	oddeven =! oddeven;	//switch phase of sorting

	}

	__syncthreads();

	//Store this phase's in[] array to out[] array
	int idx = threadIdx.x;

	if ( idx < size )
		out[idx] = in[idx];
		
}


int main(void)
{
	int i;
	int *a, *a_sorted;
	int *d_a, *d_sorted;
	int n = 6;		//make sure to keep this even
	int size = sizeof(int) *n;

/*
-----Why double pointer in void?-----
All CUDA API functions return an error code (or cudaSuccess if no error occured). All other parameters are passed by reference. However, in plain C you cannot have references, that's why you have to pass an address of the variable that you want the return information to be stored. Since you are returning a pointer, you need to pass a double-pointer.
-------------------------------------

-----cudaMalloc------
Allocates size bytes of linear memory on the device (GPU)
---------------------
*/

	hipMalloc( (void**) &d_a, size);
	hipMalloc( (void**) &d_sorted, size);

	a = (int*) malloc(size);
	a_sorted = (int*) malloc(size);

	printf("Enter the unsorted numbers:\n");
	
	for(i=0;i<n;i++){
		scanf("%d",&a[i]);
	}
	
	//d_a -> destination. a -> source.
	//Host to device array copy
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	
	//<<< >>> CUDA semantic
	testKernel<<<1,n>>>(d_a, d_sorted, n);

	//Device to Host array for final display (I/O)
	hipMemcpy(a_sorted, d_sorted, size, hipMemcpyDeviceToHost);
	
	for (i=0;i<n;i++){
		printf("%d\t",a_sorted[i]);
	}
	
	printf("\n");
	
	//free memory allocated by malloc and cudamalloc
	free(a);
	free(a_sorted);
	hipFree(d_sorted);
	hipFree(d_a);
}

